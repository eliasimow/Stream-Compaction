#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        //int blockSize = 256;

        __global__ void upScan(int n, int* data, int levelPowered) {
            if ((blockIdx.x * blockDim.x) + threadIdx.x > n) {
                return;
            }
            int stride = (1 << levelPowered);
            int groups = n / (1 << levelPowered);
            int index = ((blockIdx.x * blockDim.x) + threadIdx.x) * (1 << levelPowered) -1;
            if (index >  n || index < 0) {
                return;
            }
            data[index] = data[index] + data[index - (1 << (levelPowered-1))];
        }

        __global__ void downSweep(int n, int* data, int levelPowered) {
            if ((blockIdx.x * blockDim.x) + threadIdx.x > n) {
                return;
            }
            int index = ((blockIdx.x * blockDim.x) + threadIdx.x) * (1 << levelPowered) - 1;
            if (index > n || index < 0) {
                return;
            }
            int store = data[index];
            data[index] = data[index - (1 << (levelPowered - 1))] + data[index];
            data[index - (1 << (levelPowered - 1))] = store;          
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool time, int blockSize) {
            // TODO
            int requiredLevels = ilog2ceil(n);
            int fullSize = (1 << requiredLevels);
            int dynamicBlockSize = 2;

            int* data;
            hipMalloc((void**)&data, fullSize * sizeof(int));
            hipMemcpy(data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            int levelPowered = 1;
            dim3 fullBlocksPerGrid(fullSize / (1 << levelPowered) / blockSize);

            if (time)
                timer().startGpuTimer();

            int runcount = 0;
            while (levelPowered <= requiredLevels) {
                fullBlocksPerGrid = dim3((fullSize / (1 << levelPowered) + blockSize) / blockSize);
                upScan << < fullBlocksPerGrid, blockSize >> > (fullSize-1, data, levelPowered);
                runcount++;
                levelPowered++;
            }

            if (time) {
                timer().endGpuTimer();
            }

            
            levelPowered--;
            fullBlocksPerGrid = dim3(1, 1, 1);
            hipMemset(data + fullSize - 1, 0, sizeof(int));



            while (levelPowered > 0) {
                fullBlocksPerGrid = dim3((fullSize / (1 << levelPowered) + blockSize) / blockSize);
                downSweep << < fullBlocksPerGrid, blockSize >> > (fullSize-1, data, levelPowered);
                levelPowered--;
            }
         
            hipMemcpy(odata, data, n * sizeof(int), hipMemcpyDeviceToHost);     

            hipFree(data);
        }



        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata, int blockSize) {
            // TODO           
            int* boolsGpu;
            int* odataGpu;
            int* scanGpu;
            int* dataGpu;

            int requiredLevels = ilog2ceil(n);
            int fullSize = (1 << requiredLevels);
            hipMalloc((void**)&boolsGpu, sizeof(int) * fullSize);
            hipMalloc((void**)&scanGpu, sizeof(int) * fullSize);
            hipMalloc((void**)&odataGpu, sizeof(int) * n);
            hipMalloc((void**)&dataGpu, sizeof(int) * n);

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();

            Common::kernResetIntBuffer << < fullBlocksPerGrid, blockSize >> > (n, dataGpu, 0);
            Common::kernResetIntBuffer << < fullBlocksPerGrid, blockSize >> > (n, odataGpu, 0);

            hipMemcpy(dataGpu, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            Common::kernMapToBoolean << < fullBlocksPerGrid, blockSize >> > (fullSize, boolsGpu, dataGpu);

            ///SCAN:

                int levelPowered = 1;
                hipMemcpy(scanGpu, boolsGpu, sizeof(int) * fullSize, hipMemcpyDeviceToDevice);

                while (levelPowered <= requiredLevels) {
                    fullBlocksPerGrid = dim3((fullSize / (1 << levelPowered) + blockSize) / blockSize);
                    upScan << < fullBlocksPerGrid, blockSize >> > (fullSize - 1, scanGpu, levelPowered);
                    levelPowered++;
                }

                levelPowered--;
                fullBlocksPerGrid = dim3(1, 1, 1);
                hipMemset(scanGpu + fullSize - 1, 0, sizeof(int));

                while (levelPowered > 0) {
                    fullBlocksPerGrid = dim3((fullSize / (1 << levelPowered) + blockSize) / blockSize);
                    downSweep << < fullBlocksPerGrid, blockSize >> > (fullSize - 1, scanGpu, levelPowered);
                    levelPowered--;
                }

            hipMemcpy(odata, scanGpu, n * sizeof(int), hipMemcpyDeviceToHost);

            //0 check for last element: this causes headaches if you just return store!
            int store = idata[n - 1] == 0 ? odata[n - 1] : odata[n - 1] + 1;
            fullBlocksPerGrid = dim3((n + blockSize - 1) / blockSize);

            Common::kernScatter << < fullBlocksPerGrid, blockSize >> > (n, odataGpu, dataGpu, boolsGpu, scanGpu);
            timer().endGpuTimer();

            hipMemcpy(odata, odataGpu, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(boolsGpu);
            hipFree(scanGpu);
            hipFree(odataGpu);
            hipFree(dataGpu);

            return store;
        }
    }
}
